#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <float.h>
#include <omp.h>
#include <limits>
#include <chrono>
#include <string>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include "mmio.h"
#include "sparse_helper.h"

using std::cout;
using std::endl;
using std::min;
using std::string;

#define CHECK_CUDA(func)                                               \
	{                                                                  \
		hipError_t status = (func);                                   \
		if (status != hipSuccess)                                     \
		{                                                              \
			printf("CUDA API failed at line %d with error: %s (%d)\n", \
				   __LINE__, hipGetErrorString(status), status);      \
			return EXIT_FAILURE;                                       \
		}                                                              \
	}

#define CHECK_CUSPARSE(func)                                               \
	{                                                                      \
		hipsparseStatus_t status = (func);                                  \
		if (status != HIPSPARSE_STATUS_SUCCESS)                             \
		{                                                                  \
			printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
				   __LINE__, hipsparseGetErrorString(status), status);      \
			return EXIT_FAILURE;                                           \
		}                                                                  \
	}

#define CHECK_CUBLAS(func)                                          \
	{                                                               \
		hipblasStatus_t status = (func);                             \
		if (status != HIPBLAS_STATUS_SUCCESS)                        \
		{                                                           \
			printf("CUBLAS API failed at line %d with error: %d\n", \
				   __LINE__, status);                               \
			return EXIT_FAILURE;                                    \
		}                                                           \
	}

struct GPUTimer
{
	GPUTimer()
	{
		hipEventCreate(&start_);
		hipEventCreate(&stop_);
		hipEventRecord(start_, 0);
	}

	~GPUTimer()
	{
		hipEventDestroy(start_);
		hipEventDestroy(stop_);
	}

	void start()
	{
		hipEventRecord(start_, 0);
	}

	float seconds()
	{
		hipEventRecord(stop_, 0);
		hipEventSynchronize(stop_);
		float time;
		hipEventElapsedTime(&time, start_, stop_);
		return time * 1e-3;
	}

private:
	hipEvent_t start_, stop_;
};

int BICG_solver_gpu(
	int n,
	int nnz,
	int n_ite,
	double &process_time,
	int &n_ite_take,
	int *csrRowPtr,
	int *csrColIndex,
	double *csrVal,
	double *rhs,
	double *x0,
	bool p_f = false,
	double *x_final = NULL)
{

	GPUTimer timer;
	double p_time = 0.0;
	bool print_flag = (n_ite > 0) & p_f;
	process_time = 0.0;

	int *d_col, *d_row;
	double *d_val, *d_x;
	double *d_r, *d_p, *d_Ap;
	double *d_r0_bar, *d_v, *d_s, *d_t;
	double *d_Ax, *d_h;
	double r1;
	int k;
	double alpha, beta, r0, dot;

	alpha = 1.0;
	const double neg_alpha = -1.0;
	beta = 0.0;
	r0 = 0.0;

	double alpha_h = 1.0;
	double beta_h = 0.0;
	double omega_h = 1.0;
	double rho_h = 1.0;
	double rho_new_h = 1.0;
	double res_h = 0.0;

	const double one_const = 1.0;
	const double zero_const = 0.0;
	const vector<double> vec_zero(n, 0.0);

	timer.start();

	CHECK_CUDA(hipMalloc((void **)&d_col, nnz * sizeof(int)));
	CHECK_CUDA(hipMalloc((void **)&d_row, (n + 1) * sizeof(int)));
	CHECK_CUDA(hipMalloc((void **)&d_val, nnz * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_x, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_r, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_p, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_Ax, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_Ap, n * sizeof(double)));

	CHECK_CUDA(hipMalloc((void **)&d_v, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_s, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_t, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_h, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void **)&d_r0_bar, n * sizeof(double)));

	p_time = timer.seconds();
	if (print_flag)
	{
		printf("Device memory allocation time(s): %e\n", p_time);
	}

	timer.start();

	CHECK_CUDA(hipMemcpy(d_col, csrColIndex, nnz * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_row, csrRowPtr, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_val, csrVal, nnz * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_x, x0, n * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_r, rhs, n * sizeof(double), hipMemcpyHostToDevice));

	CHECK_CUDA(hipMemcpy(d_v, vec_zero.data(), n * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_p, vec_zero.data(), n * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_s, vec_zero.data(), n * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_t, vec_zero.data(), n * sizeof(double), hipMemcpyHostToDevice));

	p_time = timer.seconds();
	if (print_flag)
	{
		printf("Host to Device memory copy time(s): %e\n", p_time);
	}

	// CUSPARSE APIs
	hipsparseHandle_t handle = NULL;
	hipsparseSpMatDescr_t matA;
	hipsparseDnVecDescr_t vecP, vecX, vecS, vecAx, vecV, vecT, vecAp;
	void *dBuffer = NULL;
	size_t bufferSize = 0;
	CHECK_CUSPARSE(hipsparseCreate(&handle))
	// Create sparse matrix A in CSR format
	CHECK_CUSPARSE(hipsparseCreateCsr(&matA, n, n, nnz,
									 d_row, d_col, d_val,
									 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
									 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
	// Create dense vectors
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecP, n, d_p, HIP_R_64F))
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, d_x, HIP_R_64F))
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecS, n, d_s, HIP_R_64F))
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecAx, n, d_Ax, HIP_R_64F))
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecV, n, d_v, HIP_R_64F))
	CHECK_CUSPARSE(hipsparseCreateDnVec(&vecT, n, d_t, HIP_R_64F))

	// allocate an external buffer if needed
	CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
		handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		&alpha, matA, vecP, &beta, vecAx, HIP_R_64F,
		HIPSPARSE_MV_ALG_DEFAULT, &bufferSize))
	CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

	/* Get handle to the CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	CHECK_CUBLAS(hipblasCreate(&cublasHandle))

	process_time = 0.0;

	if (print_flag)
	{
		printf("###GPU#GPU#GPU#GPU#GPU#GPU###\n");
	}

	timer.start();
	hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				 &one_const, matA, vecX, &zero_const, vecAx, HIP_R_64F,
				 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

	hipblasDaxpy(cublasHandle, n, &neg_alpha, d_Ax, 1, d_r, 1);
	hipblasDdot(cublasHandle, n, d_r, 1, d_r, 1, &r1);

	// r0 hat
	hipblasDcopy(cublasHandle, n, d_r, 1, d_r0_bar, 1);

	k = 0;

	if (print_flag)
	{
		printf("iteration = %3d, res = %e\n", k, r1);
	}

	if (n_ite < 0)
	{
		n_ite = 10;
	}

	double rr = r1;

	while (k < n_ite && rr >= 1.0e-8)
	{
		// rho_new = r0_bar' * r;
		hipblasDdot(cublasHandle, n, d_r0_bar, 1, d_r, 1, &rho_new_h);

		beta_h = (rho_new_h / rho_h) * (alpha_h / omega_h);

		// d_p = d_r + beta_h * (d_p - omega_h * d_v)
		alpha = 0.0 - omega_h;
		hipblasDaxpy(cublasHandle, n, &alpha, d_v, 1, d_p, 1); // d_p <- (- omega) * d_v + d_p
		hipblasDscal(cublasHandle, n, &beta_h, d_p, 1);		  // d_p <- d_p * beta_h
		alpha = 1.0;
		hipblasDaxpy(cublasHandle, n, &alpha, d_r, 1, d_p, 1); // d_p <- 1.0 * d_r + d_p

		// d_v = A * d_p;
		hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
					 &one_const, matA, vecP, &zero_const, vecV, HIP_R_64F,
					 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

		// alpha_h = rho_new_h / (d_r0_bar' * d_v);
		hipblasDdot(cublasHandle, n, d_r0_bar, 1, d_v, 1, &dot);
		alpha_h = rho_new_h / dot;

		// d_h = d_x + alpha_h * d_p;
		hipblasDcopy(cublasHandle, n, d_x, 1, d_h, 1); // d_h <- d_x
		alpha = alpha_h;
		hipblasDaxpy(cublasHandle, n, &alpha, d_p, 1, d_h, 1); // d_h <- (alpha_h) * d_p + d_h

		// d_s = d_r - alpha_h * d_v;
		hipblasDcopy(cublasHandle, n, d_r, 1, d_s, 1); // d_s <- d_r
		alpha = 0.0 - alpha_h;
		hipblasDaxpy(cublasHandle, n, &alpha, d_v, 1, d_s, 1); // d_s <- (- alpha_h) * d_v + d_s

		// d_t = A * d_s;
		hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
					 &one_const, matA, vecS, &zero_const, vecT, HIP_R_64F,
					 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

		// omega_h = (d_t' * d_s) / (d_t' * d_t);
		hipblasDdot(cublasHandle, n, d_t, 1, d_s, 1, &omega_h);
		hipblasDdot(cublasHandle, n, d_t, 1, d_t, 1, &dot);
		omega_h = omega_h / dot;

		// d_x = d_h + omega_h * d_s;
		hipblasDcopy(cublasHandle, n, d_h, 1, d_x, 1); // d_x <- d_h
		alpha = omega_h;
		hipblasDaxpy(cublasHandle, n, &alpha, d_s, 1, d_x, 1); // d_x <- (omega_h) * d_s + d_x

		// d_r = d_s - omega_h * d_t;
		hipblasDcopy(cublasHandle, n, d_s, 1, d_r, 1); // d_r <- d_s
		alpha = 0.0 - omega_h;
		hipblasDaxpy(cublasHandle, n, &alpha, d_t, 1, d_r, 1); // d_r <- (- omega_h) * d_t + d_r

		rho_h = rho_new_h;

		hipblasDdot(cublasHandle, n, d_r, 1, d_r, 1, &rr);

		k++;
		if (print_flag)
		{
			printf("iteration = %3d, res = %e\n", k, rr);
		}
	}

	p_time = timer.seconds();
	// printf("Compute time(s): %e\n", p_time);
	process_time = p_time;
	n_ite_take = k;

	hipsparseDestroySpMat(matA);
	hipsparseDestroyDnVec(vecP);
	hipsparseDestroyDnVec(vecX);
	hipsparseDestroyDnVec(vecS);
	hipsparseDestroyDnVec(vecAx);
	hipsparseDestroyDnVec(vecV);
	hipsparseDestroyDnVec(vecT);
	hipsparseDestroy(handle);
	hipblasDestroy(cublasHandle);

	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_Ax);
	hipFree(d_Ap);

	hipFree(d_v);
	hipFree(d_s);
	hipFree(d_t);
	hipFree(d_h);
	hipFree(d_r0_bar);

	return 0;
}

int main(int argc, char *argv[])
{
	/* */
	bool p_f = false;

	if (argc < 3)
	{
		std::cout << "Usage: ./cg [sparse matrix A] [max ite] [print?]\n";
		return -1;
	}

	char *filename_A = argv[1];
	// char * filename_output = argv[2];
	// int N = 512;
	if (argc > 3)
	{
		p_f = true;
	}

	int M, K, nnz;
	vector<int> CSRRowPtr;
	vector<int> CSRColIndex;
	vector<double> CSRVal;

	cout << "Reading sparse A matrix ...";

	read_suitsparse_matrix_FP64(filename_A,
								CSRRowPtr,
								CSRColIndex,
								CSRVal,
								M,
								K,
								nnz,
								CSR);
	assert(M == K);

	cout << "Matrix size: \n";
	cout << "A: sparse matrix, " << M << " x " << K << ". NNZ = " << nnz << "\n";

	vector<double> b(M);
	cout << "Set b to [1,1,1...,1]^T \n";
	for (int i = 0; i < M; ++i)
	{
		b[i] = 1.0;
	}

	cout << "Set x0 to [0,0,0...,0]^T \n";
	vector<double> x0(M);
	for (int i = 0; i < M; ++i)
	{
		x0[i] = 0.0;
	}

	cout << "GPU Running\n";

	int n_ite = atoi(argv[2]);

	double process_time = 0.0;
	int n_ite_take = 0;
	BICG_solver_gpu(M, nnz, 10,
					process_time,
					n_ite_take,
					CSRRowPtr.data(), CSRColIndex.data(), CSRVal.data(),
					b.data(), x0.data());

	process_time = 0.0;
	BICG_solver_gpu(M, nnz, n_ite,
					process_time,
					n_ite_take,
					CSRRowPtr.data(), CSRColIndex.data(), CSRVal.data(),
					b.data(), x0.data(), p_f);

	cout << "GPU BiCG solver time(s): " << process_time << "\n";
	cout << "Iteration number: " << n_ite_take << "\n";

	return 0;
}